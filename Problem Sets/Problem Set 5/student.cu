#include "hip/hip_runtime.h"
/* Udacity HW5
 Histogramming for Speed

 The goal of this assignment is compute a histogram
 as fast as possible.  We have simplified the problem as much as
 possible to allow you to focus solely on the histogramming algorithm.

 The input values that you need to histogram are already the exact
 bins that need to be updated.  This is unlike in HW3 where you needed
 to compute the range of the data and then do:
 bin = (val - valMin) / valRange to determine the bin.

 Here the bin is just:
 bin = val

 so the serial histogram calculation looks like:
 for (i = 0; i < numElems; ++i)
 histo[val[i]]++;

 That's it!  Your job is to make it run as fast as possible!

 The values are normally distributed - you may take
 advantage of this fact in your implementation.

 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "utils.h"

__global__
void
histoSerial (const unsigned int* const d_vals, unsigned int* const d_histo,
	     int numVals)
{
  for (int i = 0; i < numVals; i++)
    {
      d_histo[d_vals[i]] += 1;
    }
}

__global__
void
histoAtomicAdd (const unsigned int* const d_vals, unsigned int* const d_histo,
		int numVals)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= numVals)
    {
      return;
    }

  atomicAdd (&d_histo[d_vals[idx]], 1);
}

__global__
void
build_group (const unsigned int* const d_vals, unsigned int* const d_group,
	     int numVals, int binSize)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int bin_idx = blockIdx.y * blockDim.y + threadIdx.y;

  if (idx >= numVals || bin_idx >= binSize)
    {
      return;
    }

  int n_groups = (numVals + binSize - 1) / binSize;
  int bin = idx / n_groups;
//  printf("n_groups: %d, bin: %d\n", n_groups, bin);
  d_group[bin * numVals + idx] = (bin_idx == bin) ? d_vals[idx] : -1;
}

__global__
void
histoSharedMem (const unsigned int* const d_group, unsigned int* const d_histo,
		int numVals, int coarseBinSize, int numThreads)
{
  extern __shared__ unsigned int histo_sh[];

  int tx = threadIdx.x;
  int group_idx = blockIdx.x;

  // TODO out-of-range check

  int numValsPerThreads = (numVals + numThreads - 1) / numThreads;

  int idx = tx * numValsPerThreads;
  if (idx >= numVals) {
      return;
  }

  for (int i=idx; i<min(numVals, idx + numValsPerThreads); i++)
    {
       int val = d_group[group_idx * numVals + i];
       if (val >= 0)
	 {
	   atomicAdd(&histo_sh[val - group_idx * coarseBinSize], 1);
	 }
    }
  __syncthreads();

  if (tx == 0)
    {
      for (int i=0; i<coarseBinSize; i++)
	{
  	  d_histo[group_idx * coarseBinSize + i] = histo_sh[i];
	}
    }
}

void
computeHistogram (const unsigned int* const d_vals, //INPUT
    unsigned int* const d_histo,      //OUTPUT
    const unsigned int numBins, const unsigned int numElems)
{
  int dev = 0;
  hipSetDevice (dev);

  hipDeviceProp_t devProps;
  if (hipGetDeviceProperties (&devProps, dev) == 0)
    {
      printf ("Using device %d:\n", dev);
      printf (
	  "%s; global mem: %luMB; shared mem: %luKB; max threads per block: %d\n",
	  devProps.name, devProps.totalGlobalMem / 1024 / 1024,
	  devProps.sharedMemPerBlock / 1024, devProps.maxThreadsPerBlock);
    }

  printf("# elems: %d\n", numElems);

  /* 1. serial version */
//  histoSerial <<<1, 1>>> (d_vals, d_histo, numElems);  // over 800 ms
  /* 2. using global memory and atomic add */
//  histoAtomicAdd <<<(numElems + 31) / 32, 32>>> (d_vals, d_histo, numElems); // around 3.1 ms

  /* TODO 3. using shared memory */
  const int COARSE_BIN_SIZE = 16;  // TODO experiment on the best size

  unsigned int* d_group;
  checkCudaErrors(
      hipMalloc (&d_group, sizeof(unsigned int) * COARSE_BIN_SIZE * numElems));
  build_group<<<dim3((numElems + 31) / 32, 32), dim3(32, 1)>>> (d_vals, d_group, numElems, COARSE_BIN_SIZE);

  int n_groups = (numElems + COARSE_BIN_SIZE - 1) / COARSE_BIN_SIZE;
  histoSharedMem<<<n_groups, 32>>>(d_group, d_histo, numElems, COARSE_BIN_SIZE, 32);

  /* TODO 4. considering data dist (normal dist) */

  hipDeviceSynchronize ();
  checkCudaErrors(hipGetLastError ());
}
