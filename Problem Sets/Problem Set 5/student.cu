#include "hip/hip_runtime.h"
/* Udacity HW5
 Histogramming for Speed

 The goal of this assignment is compute a histogram
 as fast as possible.  We have simplified the problem as much as
 possible to allow you to focus solely on the histogramming algorithm.

 The input values that you need to histogram are already the exact
 bins that need to be updated.  This is unlike in HW3 where you needed
 to compute the range of the data and then do:
 bin = (val - valMin) / valRange to determine the bin.

 Here the bin is just:
 bin = val

 so the serial histogram calculation looks like:
 for (i = 0; i < numElems; ++i)
 histo[val[i]]++;

 That's it!  Your job is to make it run as fast as possible!

 The values are normally distributed - you may take
 advantage of this fact in your implementation.

 */

#include "utils.h"

__global__
void
histoSerial (const unsigned int* const vals, unsigned int* const histo,
	     int numVals)
{
  for (int i = 0; i < numVals; i++)
    {
      histo[vals[i]] += 1;
    }
}

// TODO parallel code
// TODO considering data dist (normal dist)

void
computeHistogram (const unsigned int* const d_vals, //INPUT
    unsigned int* const d_histo,      //OUTPUT
    const unsigned int numBins, const unsigned int numElems)
{
  histoSerial <<<1, 1>>> (d_vals, d_histo, numElems);

  hipDeviceSynchronize ();
  checkCudaErrors(hipGetLastError ());
}
