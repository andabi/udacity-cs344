#include "hip/hip_runtime.h"
/* Udacity HW5
 Histogramming for Speed

 The goal of this assignment is compute a histogram
 as fast as possible.  We have simplified the problem as much as
 possible to allow you to focus solely on the histogramming algorithm.

 The input values that you need to histogram are already the exact
 bins that need to be updated.  This is unlike in HW3 where you needed
 to compute the range of the data and then do:
 bin = (val - valMin) / valRange to determine the bin.

 Here the bin is just:
 bin = val

 so the serial histogram calculation looks like:
 for (i = 0; i < numElems; ++i)
 histo[val[i]]++;

 That's it!  Your job is to make it run as fast as possible!

 The values are normally distributed - you may take
 advantage of this fact in your implementation.

 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "utils.h"

__global__
void
histoSerial (const unsigned int* const d_vals, unsigned int* const d_histo,
	     int numElems)
{
  for (int i = 0; i < numElems; i++)
    {
      d_histo[d_vals[i]] += 1;
    }
}

__global__
void
histoAtomicAdd (const unsigned int* const d_vals, unsigned int* const d_histo,
		int numElems)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= numElems)
      return;

  atomicAdd (&d_histo[d_vals[idx]], 1);
}

__global__
void
histoSharedAtomicAdd(const unsigned int* const d_vals, unsigned int* const d_histo, int numElems, int numBins)
{
  extern __shared__ unsigned int sh_histo[];

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= numElems)
    return;

  // initialize sh_histo
  for (int i=threadIdx.x; i<numBins; i+=blockDim.x)
    {
      sh_histo[i] = 0;
    }
  __syncthreads();

  atomicAdd (&sh_histo[d_vals[idx]], 1);
  __syncthreads();

  // sum up to d_histo
  for (int i=threadIdx.x; i<numBins; i+=blockDim.x)
    {
      atomicAdd (&d_histo[i], sh_histo[i]);
    }
}

__global__
void
build_group (const unsigned int* const d_vals, unsigned int* const d_group,
	     int numElems, int binSize)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int bin_idx = blockIdx.y * blockDim.y + threadIdx.y;

  if (idx >= numElems || bin_idx >= binSize)
      return;

  int n_groups = (numElems + binSize - 1) / binSize;
  int bin = idx / n_groups;
//  printf("n_groups: %d, bin: %d\n", n_groups, bin);
  d_group[bin * numElems + idx] = (bin_idx == bin) ? d_vals[idx] : -1;
}

__global__
void
histoShared (const unsigned int* const d_group, unsigned int* const d_histo,
		int numElems, int coarseBinSize, int numThreads)
{
  extern __shared__ unsigned int histo_sh[];

  int tx = threadIdx.x;
  int group_idx = blockIdx.x;

  // TODO out-of-range check

  int numElemsPerThreads = (numElems + numThreads - 1) / numThreads;

  int idx = tx * numElemsPerThreads;
  if (idx >= numElems)
      return;

  for (int i=idx; i<min(numElems, idx + numElemsPerThreads); i++)
    {
       int val = d_group[group_idx * numElems + i];
       if (val >= 0)
	 {
	   atomicAdd(&histo_sh[val - group_idx * coarseBinSize], 1);
	 }
    }
  __syncthreads();

  if (tx == 0)
    {
      for (int i=0; i<coarseBinSize; i++)
	{
  	  d_histo[group_idx * coarseBinSize + i] = histo_sh[i];
	}
    }
}

void
computeHistogram (const unsigned int* const d_vals, //INPUT
    unsigned int* const d_histo,      //OUTPUT
    const unsigned int numBins, const unsigned int numElems)
{
//  int dev = 0;
//  hipSetDevice (dev);
//
//  hipDeviceProp_t devProps;
//  if (hipGetDeviceProperties (&devProps, dev) == 0)
//    {
//      printf ("Using device %d:\n", dev);
//      printf (
//	  "%s; global mem: %luMB; shared mem: %luKB; max threads per block: %d\n",
//	  devProps.name, devProps.totalGlobalMem / 1024 / 1024,
//	  devProps.sharedMemPerBlock / 1024, devProps.maxThreadsPerBlock);
//    }
//
//  printf("# elems: %d, # bins: %d\n", numElems, numBins);

  /* serial version */
//  histoSerial <<<1, 1>>> (d_vals, d_histo, numElems);
  /* perf: over 800 ms */

  /* using global memory and atomic add */
//  const int N_THREADS = 1024;
//  histoAtomicAdd <<<(numElems + N_THREADS - 1) / N_THREADS, N_THREADS>>> (d_vals, d_histo, numElems);
  /* perf: around 3.1 ms */

  /* using shared memory and atomic add */
//  const int N_THREADS = 1024;
//  histoSharedAtomicAdd <<<(numElems + N_THREADS - 1) / N_THREADS, N_THREADS, sizeof(unsigned int) * numBins>>> (d_vals, d_histo, numElems, numBins);
  /* perf: around 0.19 ms */

  /* TODO using shared memory */
//  const int COARSE_BIN_SIZE = 16;  // TODO experiment on the best size
//
//  unsigned int* d_group;
//  checkCudaErrors(
//      hipMalloc (&d_group, sizeof(unsigned int) * COARSE_BIN_SIZE * numElems));
//  build_group<<<dim3((numElems + 31) / 32, 32), dim3(32, 1)>>> (d_vals, d_group, numElems, COARSE_BIN_SIZE);
//
//  int n_groups = (numElems + COARSE_BIN_SIZE - 1) / COARSE_BIN_SIZE;
//  histoShared<<<n_groups, 32>>>(d_group, d_histo, numElems, COARSE_BIN_SIZE, 32);

  /* TODO considering data dist (normal dist) */

  hipDeviceSynchronize ();
  checkCudaErrors(hipGetLastError ());
}
