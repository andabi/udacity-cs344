#include "hip/hip_runtime.h"
/* Udacity Homework 3
 HDR Tone-mapping

 Background HDR
 ==============

 A High Dynamic Range (HDR) image contains a wider variation of intensity
 and color than is allowed by the RGB format with 1 byte per channel that we
 have used in the previous assignment.

 To store this extra information we use single precision floating point for
 each channel.  This allows for an extremely wide range of intensity values.

 In the image for this assignment, the inside of church with light coming in
 through stained glass windows, the raw input floating point values for the
 channels range from 0 to 275.  But the mean is .41 and 98% of the values are
 less than 3!  This means that certain areas (the windows) are extremely bright
 compared to everywhere else.  If we linearly map this [0-275] range into the
 [0-255] range that we have been using then most values will be mapped to zero!
 The only thing we will be able to see are the very brightest areas - the
 windows - everything else will appear pitch black.

 The problem is that although we have cameras capable of recording the wide
 range of intensity that exists in the real world our monitors are not capable
 of displaying them.  Our eyes are also quite capable of observing a much wider
 range of intensities than our image formats / monitors are capable of
 displaying.

 Tone-mapping is a process that transforms the intensities in the image so that
 the brightest values aren't nearly so far away from the mean.  That way when
 we transform the values into [0-255] we can actually see the entire image.
 There are many ways to perform this process and it is as much an art as a
 science - there is no single "right" answer.  In this homework we will
 implement one possible technique.

 Background Chrominance-Luminance
 ================================

 The RGB space that we have been using to represent images can be thought of as
 one possible set of axes spanning a three dimensional space of color.  We
 sometimes choose other axes to represent this space because they make certain
 operations more convenient.

 Another possible way of representing a color image is to separate the color
 information (chromaticity) from the brightness information.  There are
 multiple different methods for doing this - a common one during the analog
 television days was known as Chrominance-Luminance or YUV.

 We choose to represent the image in this way so that we can remap only the
 intensity channel and then recombine the new intensity values with the color
 information to form the final image.

 Old TV signals used to be transmitted in this way so that black & white
 televisions could display the luminance channel while color televisions would
 display all three of the channels.


 Tone-mapping
 ============

 In this assignment we are going to transform the luminance channel (actually
 the log of the luminance, but this is unimportant for the parts of the
 algorithm that you will be implementing) by compressing its range to [0, 1].
 To do this we need the cumulative distribution of the luminance values.

 Example
 -------

 input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
 min / max / range: 0 / 9 / 9

 histo with 3 bins: [4 7 3]

 cdf : [4 11 14]


 Your task is to calculate this cumulative distribution by following these
 steps.

 */

#include "utils.h"

__global__ void min_step(const float* const d_input,
                         float* const d_output,
                         const int idx_steps,
                         const int n_threads){

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // out-of-bound check
  if (idx >= n_threads) {
      return;
  }

  const int n_elems = sizeof(d_input) / sizeof(float);
  if (n_elems <= idx + pow(2, idx_steps)) {
      d_output[idx] = d_input[idx];
  } else {
    d_output[idx] = min(d_input[idx], d_input[idx + pow(2, idx_steps)]);
  }
}

float
get_min (const float* const d_logLuminance)
{
  const int n_elems = sizeof(d_logLuminance)/sizeof(float);
  assert (n_elems > 0);

  float* d_output;
  checkCudaError(hipMalloc(&d_output, sizeof(float) * (int)ceil(n_elems/2)));

  const dim3 blockSize(32);

  const int n_steps = ceil (log2 (n_elems));
  float* d_input = d_logLuminance;
  for (int i = 0; i < n_steps; i++)
    {
      int n_threads = ceil(n_elems/pow(2,i+1));
      dim3 gridSize(n_threads/blockSize.x + 1);
      min_step<<<gridSize, blockSize>>>(d_input, d_output, i, n_threads);
      d_input = d_output;
    }
  return d_output[0];
}

void
your_histogram_and_prefixsum (const float* const d_logLuminance,
			      unsigned int* const d_cdf, float &min_logLum,
			      float &max_logLum, const size_t numRows,
			      const size_t numCols, const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
   1) find the minimum and maximum value in the input logLuminance channel
   store in min_logLum and max_logLum
   2) subtract them to find the range
   3) generate a histogram of all the values in the logLuminance channel using
   the formula: bin = (lum[i] - lumMin) / lumRange * numBins
   4) Perform an exclusive scan (prefix sum) on the histogram to get
   the cumulative distribution of luminance values (this should go in the
   incoming d_cdf pointer which already has been allocated for you)       */

  // find the min/max
  min_logLum = get_min(d_logLuminance);
}
