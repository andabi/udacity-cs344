#include "hip/hip_runtime.h"
/* Udacity Homework 3
 HDR Tone-mapping

 Background HDR
 ==============

 A High Dynamic Range (HDR) image contains a wider variation of intensity
 and color than is allowed by the RGB format with 1 byte per channel that we
 have used in the previous assignment.

 To store this extra information we use single precision floating point for
 each channel.  This allows for an extremely wide range of intensity values.

 In the image for this assignment, the inside of church with light coming in
 through stained glass windows, the raw input floating point values for the
 channels range from 0 to 275.  But the mean is .41 and 98% of the values are
 less than 3!  This means that certain areas (the windows) are extremely bright
 compared to everywhere else.  If we linearly map this [0-275] range into the
 [0-255] range that we have been using then most values will be mapped to zero!
 The only thing we will be able to see are the very brightest areas - the
 windows - everything else will appear pitch black.

 The problem is that although we have cameras capable of recording the wide
 range of intensity that exists in the real world our monitors are not capable
 of displaying them.  Our eyes are also quite capable of observing a much wider
 range of intensities than our image formats / monitors are capable of
 displaying.

 Tone-mapping is a process that transforms the intensities in the image so that
 the brightest values aren't nearly so far away from the mean.  That way when
 we transform the values into [0-255] we can actually see the entire image.
 There are many ways to perform this process and it is as much an art as a
 science - there is no single "right" answer.  In this homework we will
 implement one possible technique.

 Background Chrominance-Luminance
 ================================

 The RGB space that we have been using to represent images can be thought of as
 one possible set of axes spanning a three dimensional space of color.  We
 sometimes choose other axes to represent this space because they make certain
 operations more convenient.

 Another possible way of representing a color image is to separate the color
 information (chromaticity) from the brightness information.  There are
 multiple different methods for doing this - a common one during the analog
 television days was known as Chrominance-Luminance or YUV.

 We choose to represent the image in this way so that we can remap only the
 intensity channel and then recombine the new intensity values with the color
 information to form the final image.

 Old TV signals used to be transmitted in this way so that black & white
 televisions could display the luminance channel while color televisions would
 display all three of the channels.


 Tone-mapping
 ============

 In this assignment we are going to transform the luminance channel (actually
 the log of the luminance, but this is unimportant for the parts of the
 algorithm that you will be implementing) by compressing its range to [0, 1].
 To do this we need the cumulative distribution of the luminance values.

 Example
 -------

 input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
 min / max / range: 0 / 9 / 9

 histo with 3 bins: [4 7 3]

 cdf : [4 11 14]


 Your task is to calculate this cumulative distribution by following these
 steps.

 */

#include "utils.h"
#include "stdio.h"

using namespace std;

template<typename T>
  void
  print_device_data (const T* const d_data, const size_t numElem)
  {
    T *h_data = (T*) malloc (sizeof(T) * numElem);
    checkCudaErrors(
	hipMemcpy (h_data, d_data, sizeof(T) * numElem,
		    hipMemcpyDeviceToHost));
    for (size_t i = 0; i < numElem; i++)
      {
	cout << h_data[i] << " ";
      }
    cout << endl;
  }

__global__ void
reduce_step (const float* const d_input, const int numPixels,
	     float* const d_output, const int stride, const int n_threads,
	     const bool is_min)
{

  const int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  // out-of-bound check
  if (thread_idx >= n_threads)
    {
      return;
    }

  const int idx = thread_idx * 2 * stride;

//  printf("%d, %d, %d, %d\n", n_threads, idx, stride, numPixels);

  if (numPixels > idx + stride)
    {
      if (is_min)
	{
	  d_output[idx] = min (d_input[idx], d_input[idx + stride]);
	}
      else
	{
	  d_output[idx] = max (d_input[idx], d_input[idx + stride]);
	}
    }
  else
    {
      d_output[idx] = d_input[idx];
    }
}

void
reduce_sequential (const float* const h_input, const size_t numPixels,
		   float &h_output, const bool is_min)
{
  assert(numPixels > 0);

  h_output = h_input[0];
  for (size_t i = 1; i < numPixels; i++)
    {
      if (is_min)
	{
	  h_output = min (h_output, h_input[i]);
	}
      else
	{
	  h_output = max (h_output, h_input[i]);
	}
    }
}

void
reduce (const float* const d_input, const size_t numPixels,
	float* const d_output, const bool is_min)
{
  assert(numPixels > 0);

  const int n_steps = ceil (log2 ((float) numPixels));
  const float* d_temp = d_input;

  for (int i = 0; i < n_steps; i++)
    {
      int n_threads = ceil (numPixels / pow (2, i + 1));

      const dim3 blockSize (32);
      dim3 gridSize (n_threads / blockSize.x + 1);

      const int stride = pow (2, i);
      reduce_step <<<gridSize, blockSize>>> (d_temp, numPixels, d_output,
					     stride, n_threads, is_min);
      d_temp = d_output;
    }
}

void
find_range (const float* const d_logLuminance, const size_t numPixels,
	    float &min_logLum, float &max_logLum, bool is_reference = false)
{
  float* d_output;
  checkCudaErrors(hipMalloc (&d_output, sizeof(float) * numPixels / 2));
  reduce (d_logLuminance, numPixels, d_output, true);
  checkCudaErrors(
      hipMemcpy (&min_logLum, d_output, sizeof(float),
		  hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree (d_output));

  checkCudaErrors(hipMalloc (&d_output, sizeof(float) * numPixels / 2));
  reduce (d_logLuminance, numPixels, d_output, false);
  checkCudaErrors(
      hipMemcpy (&max_logLum, d_output, sizeof(float),
		  hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree (d_output));

//  cout << "min: " << min_logLum << ", max: " << max_logLum << endl;

  if (is_reference)
    {
      float *h_logLuminance = (float *) malloc (sizeof(float) * numPixels);
      float h_output_max, h_output_min;
      checkCudaErrors(
	  hipMemcpy (h_logLuminance, d_logLuminance, sizeof(float) * numPixels,
		      hipMemcpyDeviceToHost));

      reduce_sequential (h_logLuminance, numPixels, h_output_min, true);
      reduce_sequential (h_logLuminance, numPixels, h_output_max, false);

      cout << "(reference) min: " << h_output_min << ", max: " << h_output_max
	  << endl;
    }
}

__global__ void
histogram (const float* const d_logLuminance, unsigned int* const d_histogram,
	   const int numPixels, const float min_logLum, const float max_logLum,
	   const int numBins)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= numPixels)
    {
      return;
    }

  const int bin = (d_logLuminance[idx] - min_logLum) / (max_logLum - min_logLum)
      * numBins;
  atomicAdd (&d_histogram[bin], 1);
}

unsigned int*
get_histogram (const float* const d_logLuminance, const size_t numPixels,
	       const float min_logLum, const float max_logLum,
	       const size_t numBins, bool is_reference = false)
{
  unsigned int* d_histogram;
  checkCudaErrors(hipMalloc (&d_histogram, sizeof(unsigned int) * numBins));

  const dim3 blockSize (32 * 32);
  const dim3 gridSize (numPixels / blockSize.x + 1);

  histogram <<<gridSize, blockSize>>> (d_logLuminance, d_histogram, numPixels,
				       min_logLum, max_logLum, numBins);

  if (is_reference)
    {
      size_t ref_histogram[numBins] =
	{ };
      float *h_logLuminance = (float*) malloc (sizeof(float) * numPixels);

      unsigned int *h_histogram = (unsigned int*) malloc (
	  sizeof(unsigned int) * numBins);
      checkCudaErrors(
	  hipMemcpy (h_histogram, d_histogram, sizeof(unsigned int) * numBins,
		      hipMemcpyDeviceToHost));

      checkCudaErrors(
	  hipMemcpy (h_logLuminance, d_logLuminance, sizeof(float) * numPixels,
		      hipMemcpyDeviceToHost));
      for (size_t i = 0; i < numPixels; i++)
	{
	  int bin = (h_logLuminance[i] - min_logLum) / (max_logLum - min_logLum)
	      * numBins;
	  ref_histogram[bin] += 1;
	}
      cout << "(reference)" << endl;
      for (size_t i = 0; i < numBins; i++)
	{
	  cout << ref_histogram[i] << " (" << h_histogram[i] << "), ";
	}
      cout << endl;
    }

  return d_histogram;
}

__global__ void
init_scan (unsigned int* const d_histogram, unsigned int numBins,
	   unsigned int* const d_temp)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= numBins)
    {
      return;
    }
  d_temp[idx] = d_histogram[idx];
}

__global__ void
copy_scan (unsigned int* const d_temp, unsigned int* const d_cdf,
	   unsigned int numBins)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numBins)
    {
      d_cdf[idx] = d_temp[idx];
    }
}

__global__ void
reduce_scan (unsigned int* const d_input, unsigned int numElem, int stride,
	     int numThreads)
{
  const int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_idx >= numThreads)
    {
      return;
    }
  const int idx = stride * thread_idx + (stride - 1);
  d_input[idx] += d_input[idx - stride / 2];
}

__global__ void
downswipe_scan (unsigned int* const d_input, unsigned int numElem, int stride,
		int numThreads)
{
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_idx >= numThreads)
    {
      return;
    }
  thread_idx = numThreads - thread_idx - 1;
  const int idx = numElem - 2 * stride * thread_idx - 1;
  unsigned int left = d_input[idx - stride], right = d_input[idx];
  d_input[idx - stride] = right;
  d_input[idx] = left + right;
}

void
compute_exclusive_scan (unsigned int* const d_histogram, const size_t numBins,
			unsigned int* const d_cdf, bool is_reference)
{
  // initialize d_cdf
  dim3 blockSize (32);
  dim3 gridSize (numBins / blockSize.x + 1);
  init_scan <<<gridSize, blockSize>>> (d_histogram, numBins, d_cdf);

  // Blelloch exclusive scan
  int numSteps = log2 ((float) numBins);
  for (int i = 0; i < numSteps; i++)
    {
      int numThreads = pow (2, numSteps - i - 1);
      int stride = pow (2, i + 1);
      reduce_scan <<<1, numThreads>>> (d_cdf, numBins, stride, numThreads);
    }
  checkCudaErrors(hipMemset (&d_cdf[numBins - 1], 0, sizeof(unsigned int)));
  for (int i = 0; i < numSteps; i++)
    {
      int numThreads = pow (2, i);
      int stride = pow (2, numSteps - i - 1);
      downswipe_scan <<<1, numThreads>>> (d_cdf, numBins, stride, numThreads);
    }

  if (is_reference)
    {
      unsigned int* ref_histogram = (unsigned int*) malloc (
	  sizeof(unsigned int) * numBins);
      checkCudaErrors(
	  hipMemcpy (ref_histogram, d_histogram,
		      sizeof(unsigned int) * numBins, hipMemcpyDeviceToHost));

      unsigned int ref_exclusive_scan[numBins] =
	{ };
      unsigned int sum = 0;
      cout << "(reference)" << endl;
      for (size_t i = 0; i < numBins; i++)
	{
	  ref_exclusive_scan[i] = sum;
	  sum += ref_histogram[i];
	  cout << ref_exclusive_scan[i] << " ";
	}
      cout << endl;
    }
}

void
your_histogram_and_prefixsum (const float* const d_logLuminance,
			      unsigned int* const d_cdf, float &min_logLum,
			      float &max_logLum, const size_t numRows,
			      const size_t numCols, const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
   1) find the minimum and maximum value in the input logLuminance channel
   store in min_logLum and max_logLum
   2) subtract them to find the range
   3) generate a histogram of all the values in the logLuminance channel using
   the formula: bin = (lum[i] - lumMin) / lumRange * numBins
   4) Perform an exclusive scan (prefix sum) on the histogram to get
   the cumulative distribution of luminance values (this should go in the
   incoming d_cdf pointer which already has been allocated for you)       */

  const size_t numPixels = numRows * numCols;

  // find the min/max
  find_range (d_logLuminance, numPixels, min_logLum, max_logLum);

// get historgram
  unsigned int* const d_histogram = get_histogram (d_logLuminance, numPixels,
						   min_logLum, max_logLum,
						   numBins, false);
// exclusive scan
  compute_exclusive_scan (d_histogram, numBins, d_cdf, false);
//  print_device_data<unsigned int> (d_cdf, numBins);

// cleanup
  checkCudaErrors(hipFree (d_histogram));
}

