#include "hip/hip_runtime.h"
/* Udacity Homework 3
 HDR Tone-mapping

 Background HDR
 ==============

 A High Dynamic Range (HDR) image contains a wider variation of intensity
 and color than is allowed by the RGB format with 1 byte per channel that we
 have used in the previous assignment.

 To store this extra information we use single precision floating point for
 each channel.  This allows for an extremely wide range of intensity values.

 In the image for this assignment, the inside of church with light coming in
 through stained glass windows, the raw input floating point values for the
 channels range from 0 to 275.  But the mean is .41 and 98% of the values are
 less than 3!  This means that certain areas (the windows) are extremely bright
 compared to everywhere else.  If we linearly map this [0-275] range into the
 [0-255] range that we have been using then most values will be mapped to zero!
 The only thing we will be able to see are the very brightest areas - the
 windows - everything else will appear pitch black.

 The problem is that although we have cameras capable of recording the wide
 range of intensity that exists in the real world our monitors are not capable
 of displaying them.  Our eyes are also quite capable of observing a much wider
 range of intensities than our image formats / monitors are capable of
 displaying.

 Tone-mapping is a process that transforms the intensities in the image so that
 the brightest values aren't nearly so far away from the mean.  That way when
 we transform the values into [0-255] we can actually see the entire image.
 There are many ways to perform this process and it is as much an art as a
 science - there is no single "right" answer.  In this homework we will
 implement one possible technique.

 Background Chrominance-Luminance
 ================================

 The RGB space that we have been using to represent images can be thought of as
 one possible set of axes spanning a three dimensional space of color.  We
 sometimes choose other axes to represent this space because they make certain
 operations more convenient.

 Another possible way of representing a color image is to separate the color
 information (chromaticity) from the brightness information.  There are
 multiple different methods for doing this - a common one during the analog
 television days was known as Chrominance-Luminance or YUV.

 We choose to represent the image in this way so that we can remap only the
 intensity channel and then recombine the new intensity values with the color
 information to form the final image.

 Old TV signals used to be transmitted in this way so that black & white
 televisions could display the luminance channel while color televisions would
 display all three of the channels.


 Tone-mapping
 ============

 In this assignment we are going to transform the luminance channel (actually
 the log of the luminance, but this is unimportant for the parts of the
 algorithm that you will be implementing) by compressing its range to [0, 1].
 To do this we need the cumulative distribution of the luminance values.

 Example
 -------

 input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
 min / max / range: 0 / 9 / 9

 histo with 3 bins: [4 7 3]

 cdf : [4 11 14]


 Your task is to calculate this cumulative distribution by following these
 steps.

 */

#include "utils.h"
#include "stdio.h"

template<typename T>
  void
  print_device_data (const T* const d_data, const size_t numElem)
  {
    T *h_data = (T*) malloc (sizeof(T) * numElem);
    checkCudaErrors(
	hipMemcpy (h_data, d_data, sizeof(T) * numElem,
		    hipMemcpyDeviceToHost));
    for (int i = 0; i < numElem; i++)
      {
	std::cout << h_data[i] << " ";
      }
    std::cout << std::endl;
  }

__global__ void
reduce_step (const float* const d_input, const int numPixels,
	     float* const d_output, const int stride, const int n_threads,
	     const bool is_min)
{

  const int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  // out-of-bound check
  if (thread_idx >= n_threads)
    {
      return;
    }

  const int idx = thread_idx * 2 * stride;

//  printf("%d, %d, %d, %d\n", n_threads, idx, stride, numPixels);

  if (numPixels > idx + stride)
    {
      if (is_min)
	{
	  d_output[idx] = min (d_input[idx], d_input[idx + stride]);
	}
      else
	{
	  d_output[idx] = max (d_input[idx], d_input[idx + stride]);
	}
    }
  else
    {
      d_output[idx] = d_input[idx];
    }
}

void
reduce_sequential (const float* const h_input, const size_t numPixels,
		   float &h_output, const bool is_min)
{
  assert(numPixels > 0);

  h_output = h_input[0];
  for (int i = 1; i < numPixels; i++)
    {
      if (is_min)
	{
	  h_output = min (h_output, h_input[i]);
	}
      else
	{
	  h_output = max (h_output, h_input[i]);
	}
    }
}

void
reduce (const float* const d_input, const size_t numPixels,
	float* const d_output, const bool is_min)
{
  assert(numPixels > 0);

  const int n_steps = ceil (log2 ((float) numPixels));
  const float* d_temp = d_input;

  for (int i = 0; i < n_steps; i++)
    {
      int n_threads = ceil (numPixels / pow (2, i + 1));

      const dim3 blockSize (32);
      dim3 gridSize (n_threads / blockSize.x + 1);

      const int stride = pow (2, i);
      reduce_step <<<gridSize, blockSize>>> (d_temp, numPixels, d_output,
					     stride, n_threads, is_min);
      d_temp = d_output;
    }
}

void
find_range (const float* const d_logLuminance, const size_t numPixels,
	    float &min_logLum, float &max_logLum, bool is_reference = false)
{
  float* d_output;
  checkCudaErrors(hipMalloc (&d_output, sizeof(float) * numPixels / 2));
  reduce (d_logLuminance, numPixels, d_output, true);
  checkCudaErrors(
      hipMemcpy (&min_logLum, d_output, sizeof(float),
		  hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree (d_output));

  checkCudaErrors(hipMalloc (&d_output, sizeof(float) * numPixels / 2));
  reduce (d_logLuminance, numPixels, d_output, false);
  checkCudaErrors(
      hipMemcpy (&max_logLum, d_output, sizeof(float),
		  hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree (d_output));

//  std::cout << "min: " << min_logLum << ", max: " << max_logLum << std::endl;

  if (is_reference)
    {
      float *h_logLuminance = (float *) malloc (sizeof(float) * numPixels);
      float h_output_max, h_output_min;
      checkCudaErrors(
	  hipMemcpy (h_logLuminance, d_logLuminance, sizeof(float) * numPixels,
		      hipMemcpyDeviceToHost));

      reduce_sequential (h_logLuminance, numPixels, h_output_min, true);
      reduce_sequential (h_logLuminance, numPixels, h_output_max, false);

      std::cout << "(reference) min: " << h_output_min << ", max: "
	  << h_output_max << std::endl;
    }
}

__global__ void
histogram (const float* const d_logLuminance, unsigned int* const d_histogram,
	   const int numPixels, const int min_logLum, const int max_logLum,
	   const int numBins)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= numPixels)
    {
      return;
    }

  const int bin = (d_logLuminance[idx] - min_logLum) / (max_logLum - min_logLum)
      * numBins;
  atomicAdd (&d_histogram[bin], 1);
}

unsigned int*
get_histogram (const float* const d_logLuminance, const size_t numPixels,
	       const float min_logLum, const float max_logLum,
	       const size_t numBins)
{
  unsigned int* d_histogram;
  checkCudaErrors(hipMalloc (&d_histogram, sizeof(unsigned int) * numBins));

  const dim3 blockSize (32 * 32);
  const dim3 gridSize (numPixels / blockSize.x + 1);

  histogram <<<gridSize, blockSize>>> (d_logLuminance, d_histogram, numPixels,
				       min_logLum, max_logLum, numBins);

  return d_histogram;
}

void
your_histogram_and_prefixsum (const float* const d_logLuminance,
			      unsigned int* const d_cdf, float &min_logLum,
			      float &max_logLum, const size_t numRows,
			      const size_t numCols, const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
   1) find the minimum and maximum value in the input logLuminance channel
   store in min_logLum and max_logLum
   2) subtract them to find the range
   3) generate a histogram of all the values in the logLuminance channel using
   the formula: bin = (lum[i] - lumMin) / lumRange * numBins
   4) Perform an exclusive scan (prefix sum) on the histogram to get
   the cumulative distribution of luminance values (this should go in the
   incoming d_cdf pointer which already has been allocated for you)       */

  const size_t numPixels = numRows * numCols;

  // find the min/max
  find_range (d_logLuminance, numPixels, min_logLum, max_logLum);
  //  print_device_data (d_logLuminance, numPixels);

  // get historgram
  unsigned int* const d_histogram = get_histogram (d_logLuminance, numPixels,
						   min_logLum, max_logLum,
						   numBins);
  print_device_data<unsigned int> (d_histogram, numBins);

  // cleanup
  checkCudaErrors(hipFree (d_histogram));
}

